#include "hip/hip_runtime.h"
#include "memory_analysis.h"

__global__ void globalMemorySearch(int *input, int totalFound, int numElements, int numThreads)
{
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadId < numThreads)
    {
        int threadSpan = numElements / numThreads;
        int offset = threadId * threadSpan;

        for (int i = offset; i < offset + threadSpan; ++i)
        {
            input[i]++; // Increment before comparison
            if (input[i] == constant_search_value)
            {
                input[i] = 1;
            }
        }
    }
}

__global__ void sharedMemorySearch(int *input, int totalFound, int numElements, int numThreads)
{
    extern __shared__ int sharedInput[];
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int threadSpan = numElements / numThreads;
    int offset = threadId * threadSpan;

    if (threadId < numThreads)
    {
        for (int i = offset; i < offset + threadSpan; ++i)
        {
            sharedInput[i] = input[i]; // Copy to shared memory
        }
        __syncthreads(); // Synchronize threads to ensure all data is in shared memory

        for (int i = offset; i < offset + threadSpan; ++i)
        {
            sharedInput[i]++;
            if (sharedInput[i] == constant_search_value)
            {
                sharedInput[i] = 1;
            }
        }
        __syncthreads(); // Synchronize before writing back to global memory

        for (int i = offset; i < offset + threadSpan; ++i)
        {
            input[i] = sharedInput[i]; // Write back to global memory
        }
    }
}

__global__ void constantMemorySearch(int totalFound)
{
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadId < constant_num_threads)
    {
        int offset = threadId * constant_thread_span;

        for (int i = offset; i < offset + constant_thread_span; ++i)
        {
            constant_input[i]++; // Increment the value in constant memory
            if (constant_input[i] == constant_search_value)
            {
                constant_input[i] = 1; // Modify constant memory
            }
        }
    }
}

__global__ void registerMemorySearch(int *input, int totalFound, int numElements, int numThreads)
{
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadId < numThreads)
    {
        int threadSpan = numElements / numThreads;
        int offset = threadId * threadSpan;
        int register_search_value = constant_search_value; // Copy to register

        for (int i = offset; i < offset + threadSpan; ++i)
        {
            input[i]++;
            if (input[i] == register_search_value)
            {
                input[i] = 1;
            }
        }
    }
}

// This will generate an array of size numElements of random integers from 0 to 255 in pageable host memory
__host__ int* allocatePageableRandomHostMemory(int numElements)
{
    srand(time(0));
    size_t size = numElements * sizeof(int);

    // Allocate the host pinned memory input pointer B
    int *data;
    hipHostAlloc((void**)&data, size, hipHostMallocDefault);

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        data[i] = rand() % 255;
    }

    return data;
}

__host__ int * allocateDeviceMemory(int numElements)
{
    size_t size = numElements * sizeof(int);

    int *d_input = NULL;
    hipError_t err = hipMalloc(&d_input, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_input (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return d_input;
}

__host__ void copyFromHostToDevice(std::string kernelType, int *input, int numElements, int numThreads, int *d_input)
{
    size_t size = numElements * sizeof(int);
    hipError_t err;

    if (!strcmp(kernelType.c_str(), "constant"))
    {
        // constant_num_threads = numThreads;
        hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(constant_num_threads), &numThreads, sizeof(int), 0, hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy num threads from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // constant_num_elements = numElements; // Copy numElements to constant memory
        err = hipMemcpyToSymbol(HIP_SYMBOL(constant_num_elements), &numElements, sizeof(int), 0, hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy num elements from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // Copy the input data to constant memory
        size_t constant_size = numElements * sizeof(int);
        err = hipMemcpy(constant_input, input, constant_size, hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy array input from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        int threadSpan = numElements / numThreads;
        err = hipMemcpyToSymbol(HIP_SYMBOL(constant_thread_span), &threadSpan, sizeof(int), 0, hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy thread span from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }

    err = hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy array input from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    
    // size_t size = numElements * sizeof(int);

    // if(!strcmp(kernelType.c_str(), "constant"))
    // {
    //     // Copy input, numElements, threadSpan, and numThreads to constant memory
    //     int threadSpan = numElements / numThreads;
    // } else
    // {
    //     // Copy input to d_input
    //     if (err != hipSuccess)
    //     {
    //         fprintf(stderr, "Failed to copy array input from host to device (error code %s)!\n", hipGetErrorString(err));
    //         exit(EXIT_FAILURE);
    //     }
    // }
}

__host__ void executeKernel(int *d_input, int numElements, int threadsPerBlock, std::string kernelType)
{
    int totalFound = 0;
    // Launch the search CUDA Kernel
    if (!strcmp(kernelType.c_str(), "global"))
    {
        globalMemorySearch<<<1, threadsPerBlock>>>(d_input, totalFound, numElements, threadsPerBlock); // you will need to fill in function arguments appropriately
    } else if (!strcmp(kernelType.c_str(), "constant"))
    {
        constantMemorySearch<<<1, threadsPerBlock>>>(totalFound); // you will need to fill in function arguments appropriately
    } else if (!strcmp(kernelType.c_str(), "shared"))
    {
        unsigned int_array_size = numElements * sizeof(int);
        sharedMemorySearch<<<1, threadsPerBlock, int_array_size>>>(d_input, totalFound, numElements, threadsPerBlock); // you will need to fill in function arguments appropriately
    } else {
        registerMemorySearch<<<1, threadsPerBlock>>>(d_input, totalFound, numElements, threadsPerBlock); // you will need to fill in function arguments appropriately
    }
    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch %s kernel (error code %s)!\n", kernelType.c_str(), hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();
}

// Free device global memory
__host__ void deallocateMemory(int *d_input)
{

    hipError_t err = hipFree(d_input);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_input (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

}

// Reset the device and exit
__host__ void cleanUpDevice()
{
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipError_t err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


__host__ std::tuple<int, std::string, int, std::string> parseCommandLineArguments(int argc, char *argv[])
{
    int elementsPerThread = 2;
    int threadsPerBlock = 256;
    std::string currentPartId = "test";
    std::string kernelType = "global";

    for(int i = 1; i < argc; i++)
    {
        std::string option(argv[i]);
        i++;
        std::string value(argv[i]);
        if(option.compare("-t") == 0) 
        {
            threadsPerBlock = atoi(value.c_str());
        }
        else if(option.compare("-m") == 0) 
        {
            elementsPerThread = atoi(value.c_str());
        }
        else if(option.compare("-p") == 0) 
        {
            currentPartId = value;
        }
        else if(option.compare("-k") == 0) 
        {
            kernelType = value;
        }
    }

    return {elementsPerThread, currentPartId, threadsPerBlock, kernelType};
}

__host__ int * setUpInput(int numElements)
{
    srand(time(0));
    int *input;

    int searchValue = rand() % 255;
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(constant_search_value), &searchValue, sizeof(int), 0, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy constant int d_v from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    input = allocatePageableRandomHostMemory(numElements);

    return input;
}

/*
 * Host main routine
 * -m elementsPerThread - the number of elements that a thread will search for a random value in
 * -p currentPartId - the Coursera Part ID
 * -t threadsPerBlock - the number of threads to schedule for concurrent processing
 * -k the kernel type - global, constant, shared, register
 */
int main(int argc, char *argv[])
{
    auto[elementsPerThread, currentPartId, threadsPerBlock, kernelType] = parseCommandLineArguments(argc, argv);

    int numElements = elementsPerThread * threadsPerBlock;
    if (numElements > 16384)
    {
        elementsPerThread = 16384 / threadsPerBlock;
        numElements = elementsPerThread * threadsPerBlock;
        std::cout << "Adjusted elements per thread to " << elementsPerThread << " to keep total elements under 16K." << std::endl;
    }

    int *input = setUpInput(numElements);
    int *d_input = allocateDeviceMemory(numElements);

    copyFromHostToDevice(kernelType, input, numElements, threadsPerBlock, d_input);

    // Start time including kernel processing time
    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventRecord(start,0);

    executeKernel(d_input, numElements, threadsPerBlock, kernelType);

    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);

    deallocateMemory(d_input);
    cleanUpDevice();
    
    ofstream outputfile;
    outputfile.open ("output.csv", std::ios_base::app);
    outputfile << currentPartId.c_str() << "," << kernelType.c_str() << "," << threadsPerBlock << "," << elementsPerThread << "," << elapsedTime << "\n";
    outputfile.close();

    return 0;
}